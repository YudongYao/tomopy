#include "hip/hip_runtime.h"
//  Copyright (c) 2015, UChicago Argonne, LLC. All rights reserved.
//  Copyright 2015. UChicago Argonne, LLC. This software was produced
//  under U.S. Government contract DE-AC02-06CH11357 for Argonne National
//  Laboratory (ANL), which is operated by UChicago Argonne, LLC for the
//  U.S. Department of Energy. The U.S. Government has rights to use,
//  reproduce, and distribute this software.  NEITHER THE GOVERNMENT NOR
//  UChicago Argonne, LLC MAKES ANY WARRANTY, EXPRESS OR IMPLIED, OR
//  ASSUMES ANY LIABILITY FOR THE USE OF THIS SOFTWARE.  If software is
//  modified to produce derivative works, such modified software should
//  be clearly marked, so as not to confuse it with the version available
//  from ANL.
//  Additionally, redistribution and use in source and binary forms, with
//  or without modification, are permitted provided that the following
//  conditions are met:
//      * Redistributions of source code must retain the above copyright
//        notice, this list of conditions and the following disclaimer.
//      * Redistributions in binary form must reproduce the above copyright
//        notice, this list of conditions and the following disclaimer in
//        the documentation andwith the
//        distribution.
//      * Neither the name of UChicago Argonne, LLC, Argonne National
//        Laboratory, ANL, the U.S. Government, nor the names of its
//        contributors may be used to endorse or promote products derived
//        from this software without specific prior written permission.
//  THIS SOFTWARE IS PROVIDED BY UChicago Argonne, LLC AND CONTRIBUTORS
//  "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
//  LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
//  FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL UChicago
//  Argonne, LLC OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
//  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
//  BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
//  LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
//  CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
//  LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
//  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
//  POSSIBILITY OF SUCH DAMAGE.
//  ---------------------------------------------------------------
//   TOMOPY CUDA implementation

#include "common.hh"
#include "gpu.hh"
#include "utils.hh"

BEGIN_EXTERN_C
#include "sirt.h"
#include "utils.h"
#include "utils_cuda.h"
#include "utils_openacc.h"
#include "utils_openmp.h"
END_EXTERN_C

#include <algorithm>
#include <chrono>
#include <cstdlib>
#include <memory>
#include <numeric>

#if !defined(cast)
#    define cast static_cast
#endif

#if defined(TOMOPY_USE_NVTX)
extern nvtxEventAttributes_t nvtx_total;
extern nvtxEventAttributes_t nvtx_iteration;
extern nvtxEventAttributes_t nvtx_slice;
extern nvtxEventAttributes_t nvtx_projection;
extern nvtxEventAttributes_t nvtx_update;
extern nvtxEventAttributes_t nvtx_rotate;
#endif

//======================================================================================//

struct gpu_data
{
    typedef gpu_data this_type;

    int           m_device;
    int           m_block;
    int           m_dy;
    int           m_dt;
    int           m_dx;
    int           m_nx;
    int           m_ny;
    float*        m_rot;
    float*        m_tmp;
    float*        m_update;
    float*        m_recon;
    float*        m_data;
    int           m_num_streams = 1;
    hipStream_t* m_streams     = nullptr;

    gpu_data(int device, int dy, int dt, int dx, int nx, int ny, const float* cpu_data)
    : m_device(device)
    , m_block(GetEnv<int>("CUDA_BLOCK_SIZE", 128))
    , m_dy(dy)
    , m_dt(dt)
    , m_dx(dx)
    , m_nx(nx)
    , m_ny(ny)
    , m_rot(nullptr)
    , m_tmp(nullptr)
    , m_update(nullptr)
    , m_recon(nullptr)
    , m_data(nullptr)
    {
        cuda_set_device(m_device);
        m_streams = create_streams(m_num_streams);
        m_rot     = gpu_malloc<float>(m_nx * m_ny);
        m_tmp     = gpu_malloc<float>(m_nx * m_ny);
        m_update  = gpu_malloc<float>(m_dy * m_nx * m_ny);
        m_recon   = gpu_malloc<float>(m_dy * m_nx * m_ny);
        m_data    = gpu_malloc<float>(m_dy * m_dt * m_dx);
        hipMemcpy(m_data, cpu_data, m_dy * m_dt * m_dx * sizeof(float),
                   hipMemcpyHostToDevice);
    }

    ~gpu_data()
    {
        hipFree(m_rot);
        hipFree(m_tmp);
        hipFree(m_update);
        hipFree(m_recon);
        hipFree(m_data);
        destroy_streams(m_streams, m_num_streams);
    }

    int compute_grid(int size) const { return (size + m_block - 1) / m_block; }

    void sync(int stream_id = -1)
    {
        auto _sync = [&](int _stream_id) {
            hipStreamSynchronize(m_streams[_stream_id]);
            CUDA_CHECK_LAST_ERROR();
        };

        if(stream_id < 0)
        {
            for(int i = 0; i < m_num_streams; ++i)
                _sync(i);
        }
        else
            _sync(stream_id);
    }

    void reset()
    {
        hipMemsetAsync(m_update, 0, m_dy * m_nx * m_ny * sizeof(float), *m_streams);
    }

    void copy(const float* recon)
    {
        hipMemcpyAsync(m_recon, recon, m_dy * m_nx * m_ny * sizeof(float),
                   hipMemcpyDeviceToDevice, *m_streams);
    }

    int          device() const { return m_device; }
    int          block() const { return m_block; }
    float*       rot() const { return m_rot; }
    float*       tmp() const { return m_tmp; }
    float*       update() const { return m_update; }
    float*       recon() const { return m_recon; }
    float*       data() const { return m_data; }
    hipStream_t stream(int stream_id = 0)
    {
        return m_streams[stream_id % m_num_streams];
    }
};

//======================================================================================//

__global__ void
cuda_sirt_sum_kernel(float* dst, const float* src, int size, const float factor)
{
    int i0      = blockIdx.x * blockDim.x + threadIdx.x;
    int istride = blockDim.x * gridDim.x;

    for(int i = i0; i < size; i += istride)
        dst[i] += factor * src[i];
}

//======================================================================================//

__global__ void
cuda_sirt_atomic_sum_kernel(float* dst, const float* src, int size, const float factor)
{
    int i0      = blockIdx.x * blockDim.x + threadIdx.x;
    int istride = blockDim.x * gridDim.x;

    for(int i = i0; i < size; i += istride)
        atomicAdd(&dst[i], factor * src[i]);
}

//======================================================================================//

__global__ void
cuda_sirt_pixels_kernel(int p, int nx, int dx, float* recon, const float* data,
                        float* dst)
{
    int i0      = blockIdx.x * blockDim.x + threadIdx.x;
    int istride = blockDim.x * gridDim.x;

    for(int d = i0; d < dx; d += istride)
    {
        int   pix_offset = d * nx;      // pixel offset
        int   idx_data   = d + p * dx;  // data offset
        float sum        = 0.0f;
        for(int i = 0; i < nx; ++i)
            sum += recon[i + pix_offset];
        float upd = (data[idx_data] - sum) / static_cast<float>(nx);
        for(int i = 0; i < nx; ++i)
            dst[i + pix_offset] += upd;
    }
}

//======================================================================================//

void
cuda_compute_projection(int dt, int dx, int nx, int ny, const float* theta, int s, int p,
                        int nthreads, gpu_data** _gpu_data)
{
    auto       thread_number = GetThisThreadID() % nthreads;
    gpu_data*& _cache        = _gpu_data[thread_number];

    hipStream_t stream = _cache->stream(0);
    cuda_set_device(_cache->device());

#if defined(DEBUG)
    printf("[%lu] Running slice %i, projection %i on device %i...\n", GetThisThreadID(),
           s, p, _cache->device());
#endif

    // needed for recon to output at proper orientation
    float theta_p_rad = fmodf(theta[p] + halfpi, twopi);
    float theta_p_deg = theta_p_rad * degrees;
    int   block       = _cache->block();
    int   grid        = _cache->compute_grid(dx);
    int   smem        = 0;

    const float* recon     = _cache->recon() + s * nx * ny;
    const float* data      = _cache->data() + s * dt * dx;
    float*       update    = _cache->update() + s * nx * ny;
    float*       recon_rot = _cache->rot();
    float*       recon_tmp = _cache->tmp();

    // Rotate object
    hipMemsetAsync(recon_rot, 0, nx * ny * sizeof(float), stream);
    CUDA_CHECK_LAST_ERROR();

    cuda_rotate_ip(recon_rot, recon, -theta_p_rad, -theta_p_deg, nx, ny, stream);
    CUDA_CHECK_LAST_ERROR();

    NVTX_RANGE_PUSH(&nvtx_update);
    cuda_sirt_pixels_kernel<<<grid, block, smem, stream>>>(p, nx, dx, recon_rot, data,
                                                           recon_rot);
    CUDA_CHECK_LAST_ERROR();
    NVTX_RANGE_POP(stream);

    // Back-Rotate object
    hipMemsetAsync(recon_tmp, 0, nx * ny * sizeof(float), stream);
    CUDA_CHECK_LAST_ERROR();

    cuda_rotate_ip(recon_tmp, recon_rot, theta_p_rad, theta_p_deg, nx, ny, stream);
    CUDA_CHECK_LAST_ERROR();

    // update shared update array
    float factor = 1.0f / static_cast<float>(dx);
    cuda_sirt_atomic_sum_kernel<<<grid, block, smem, stream>>>(update, recon_tmp, nx * ny,
                                                               factor);
    CUDA_CHECK_LAST_ERROR();

    hipStreamSynchronize(stream);
    CUDA_CHECK_LAST_ERROR();
}

//--------------------------------------------------------------------------------------//

void
sirt_cuda(const float* cpu_data, int dy, int dt, int dx, const float* center,
          const float* theta, float* cpu_recon, int ngridx, int ngridy, int num_iter)
{
    if(cuda_device_count() == 0)
        throw std::runtime_error("No CUDA device(s) available");

    init_nvtx();
    cuda_device_query();

    printf("\n\t%s [nitr = %i, dy = %i, dt = %i, dx = %i, nx = %i, ny = %i]\n\n",
           __FUNCTION__, num_iter, dy, dt, dx, ngridx, ngridy);

    auto tid = GetThisThreadID();

    // get some properties
    int num_devices       = cuda_device_count();
    int nthreads          = GetEnv("TOMOPY_NUM_THREADS", 1);
    nthreads              = std::max(nthreads, 1);
    hipStream_t* streams = create_streams(num_devices);

    // assign the thread to a device
    static std::atomic<int> ntid;
    int                     thread_device = (ntid++) % num_devices;

#if defined(TOMOPY_USE_PTL)
    TaskRunManager* run_man = cpu_run_manager();
    init_run_manager(run_man, nthreads);
    TaskManager* task_man = run_man->GetTaskManager();
    ThreadPool*  tp       = task_man->thread_pool();
#endif

    TIMEMORY_AUTO_TIMER("");

    // GPU allocated copies
    cuda_set_device(thread_device);
    printf("[%lu] Running on device %i...\n", GetThisThreadID(), thread_device);

    float* tmp_recon = gpu_malloc<float>(dy * ngridx * ngridy);
    float* recon     = gpu_malloc<float>(dy * ngridx * ngridy);
    hipMemcpy(recon, cpu_recon, dy * ngridx * ngridy * sizeof(float),
               hipMemcpyHostToDevice);
    gpu_data** _gpu_data = new gpu_data*[nthreads];

    for(int ii = 0; ii < nthreads; ++ii)
        _gpu_data[ii] = new gpu_data(thread_device, dy, dt, dx, ngridx, ngridy, cpu_data);

    NVTX_RANGE_PUSH(&nvtx_total);

    for(int i = 0; i < num_iter; i++)
    {
        NVTX_RANGE_PUSH(&nvtx_iteration);
        START_TIMER(t_start);

        // set "update" to zero, copy in "recon"
        for(int ii = 0; ii < nthreads; ++ii)
        {
            _gpu_data[ii]->reset();
            _gpu_data[ii]->copy(recon);
        }

        // Loop over slices
        for(int s = 0; s < dy; ++s)
        {
            NVTX_RANGE_PUSH(&nvtx_slice);

#if defined(TOMOPY_USE_PTL)
            TaskGroup<void> tg;
            // For each projection angle
            for(int p = 0; p < dt; p++)
                task_man->exec(tg, cuda_compute_projection, dt, dx, ngridx, ngridy, theta,
                               s, p, nthreads, _gpu_data);
            tg.join();
#else
            // For each projection angle
            for(int p = 0; p < dt; p++)
                cuda_compute_projection(dt, dx, ngridx, ngridy, theta, s, p, nthreads,
                                        _gpu_data);
#endif
            NVTX_RANGE_POP(0);
        }

        for(int ii = 0; ii < nthreads; ++ii)
        {
            int    block  = _gpu_data[ii]->block();
            int    grid   = _gpu_data[ii]->compute_grid(dy * ngridx * ngridy);
            float* update = _gpu_data[ii]->update();
            hipStream_t stream = _gpu_data[ii]->stream();
            float  factor = 1.0f;
            cuda_sirt_atomic_sum_kernel<<<grid, block, 0, stream>>>(recon, update, dy * ngridx * ngridy,
                                                  factor);
        }
        REPORT_TIMER(t_start, "iteration", i, num_iter);
        NVTX_RANGE_POP(0);
    }
    printf("\n");

    hipDeviceSynchronize();
    hipMemcpy(cpu_recon, recon, dy * ngridx * ngridy * sizeof(float),
               hipMemcpyDeviceToHost);
    hipFree(recon);
    hipFree(tmp_recon);

    for(int i = 0; i < nthreads; ++i)
        delete _gpu_data[i];
    delete[] _gpu_data;

    hipDeviceSynchronize();
    destroy_streams(streams, num_devices);
    NVTX_RANGE_POP(0);
}

//======================================================================================//
