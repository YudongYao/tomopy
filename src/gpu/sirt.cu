#include "hip/hip_runtime.h"
//  Copyright (c) 2015, UChicago Argonne, LLC. All rights reserved.
//  Copyright 2015. UChicago Argonne, LLC. This software was produced
//  under U.S. Government contract DE-AC02-06CH11357 for Argonne National
//  Laboratory (ANL), which is operated by UChicago Argonne, LLC for the
//  U.S. Department of Energy. The U.S. Government has rights to use,
//  reproduce, and distribute this software.  NEITHER THE GOVERNMENT NOR
//  UChicago Argonne, LLC MAKES ANY WARRANTY, EXPRESS OR IMPLIED, OR
//  ASSUMES ANY LIABILITY FOR THE USE OF THIS SOFTWARE.  If software is
//  modified to produce derivative works, such modified software should
//  be clearly marked, so as not to confuse it with the version available
//  from ANL.
//  Additionally, redistribution and use in source and binary forms, with
//  or without modification, are permitted provided that the following
//  conditions are met:
//      * Redistributions of source code must retain the above copyright
//        notice, this list of conditions and the following disclaimer.
//      * Redistributions in binary form must reproduce the above copyright
//        notice, this list of conditions and the following disclaimer in
//        the documentation andwith the
//        distribution.
//      * Neither the name of UChicago Argonne, LLC, Argonne National
//        Laboratory, ANL, the U.S. Government, nor the names of its
//        contributors may be used to endorse or promote products derived
//        from this software without specific prior written permission.
//  THIS SOFTWARE IS PROVIDED BY UChicago Argonne, LLC AND CONTRIBUTORS
//  "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
//  LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
//  FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL UChicago
//  Argonne, LLC OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
//  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
//  BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
//  LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
//  CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
//  LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
//  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
//  POSSIBILITY OF SUCH DAMAGE.
//  ---------------------------------------------------------------
//   TOMOPY CUDA implementation

#include "common.hh"
#include "gpu.hh"
#include "utils.hh"

BEGIN_EXTERN_C
#include "sirt.h"
#include "utils.h"
#include "utils_cuda.h"
#include "utils_openacc.h"
#include "utils_openmp.h"
END_EXTERN_C

#include <algorithm>
#include <chrono>
#include <cstdlib>
#include <memory>
#include <numeric>

#if !defined(cast)
#    define cast static_cast
#endif

#if defined(TOMOPY_USE_NVTX)
extern nvtxEventAttributes_t nvtx_update;
extern nvtxEventAttributes_t nvtx_rotate;
#endif

//======================================================================================//

struct gpu_data
{
    typedef gpu_data this_type;

    int    m_device;
    int    m_id;
    int    m_block;
    int    m_dy;
    int    m_dt;
    int    m_dx;
    int    m_nx;
    int    m_ny;
    float* m_rot;
    float* m_tmp;
    float* m_update;
    float* m_recon;
    float* m_data;

    gpu_data(int device, int id, int dy, int dt, int dx, int nx, int ny,
             const float* cpu_data)
    : m_device(device)
    , m_id(id)
    , m_block(GetEnv<int>("CUDA_BLOCK_SIZE", 128))
    , m_dy(dy)
    , m_dt(dt)
    , m_dx(dx)
    , m_nx(nx)
    , m_ny(ny)
    , m_rot(nullptr)
    , m_tmp(nullptr)
    , m_update(nullptr)
    , m_recon(nullptr)
    , m_data(nullptr)
    {
        cuda_set_device(m_device);
        m_rot    = gpu_malloc<float>(m_nx * m_ny);
        m_tmp    = gpu_malloc<float>(m_nx * m_ny);
        m_update = gpu_malloc<float>(m_dy * m_nx * m_ny);
        m_recon  = gpu_malloc<float>(m_dy * m_nx * m_ny);
        m_data   = gpu_malloc<float>(m_dy * m_dt * m_dx);
        hipMemcpy(m_data, cpu_data, m_dy * m_dt * m_dx * sizeof(float),
                   hipMemcpyHostToDevice);
    }

    ~gpu_data()
    {
        hipFree(m_rot);
        hipFree(m_tmp);
        hipFree(m_update);
        hipFree(m_recon);
        hipFree(m_data);
    }

    int compute_grid(int size) const { return (size + m_block - 1) / m_block; }

    void sync()
    {
        hipStreamSynchronize(0);
        CUDA_CHECK_LAST_ERROR();
    }

    void reset()
    {
        if(m_device != 0)
        {
            cuda_set_device(m_device);
            hipDeviceSynchronize();
        }
        hipMemset(m_update, 0, m_dy * m_nx * m_ny * sizeof(float));
        if(m_device != 0)
            cuda_set_device(0);
    }

    void copy(const float* recon)
    {
        if(m_device != 0)
            hipMemcpyPeer(m_recon, m_device, recon, 0,
                           m_dy * m_nx * m_ny * sizeof(float));
        else
            hipMemcpy(m_recon, recon, m_dy * m_nx * m_ny * sizeof(float),
                       hipMemcpyDeviceToDevice);
    }

    int    device() const { return m_device; }
    int    block() const { return m_block; }
    float* rot() const { return m_rot; }
    float* tmp() const { return m_tmp; }
    float* update() const { return m_update; }
    float* recon() const { return m_recon; }
    float* data() const { return m_data; }
};

//======================================================================================//

__global__ void
cuda_sirt_sum_kernel(float* dst, const float* src, int size, const float factor)
{
    int i0      = blockIdx.x * blockDim.x + threadIdx.x;
    int istride = blockDim.x * gridDim.x;

    for(int i = i0; i < size; i += istride)
        dst[i] += factor * src[i];
}

//======================================================================================//

__global__ void
cuda_sirt_atomic_sum_kernel(float* dst, const float* src, int size, const float factor)
{
    int i0      = blockIdx.x * blockDim.x + threadIdx.x;
    int istride = blockDim.x * gridDim.x;

    for(int i = i0; i < size; i += istride)
        atomicAdd(&dst[i], factor * src[i]);
}

//======================================================================================//

__global__ void
cuda_sirt_pixels_kernel(int p, int nx, int dx, float* recon, const float* data, float* dst)
{
    int i0      = blockIdx.x * blockDim.x + threadIdx.x;
    int istride = blockDim.x * gridDim.x;

    for(int d = i0; d < dx; d += istride)
    {
        int   pix_offset = d * nx;      // pixel offset
        int   idx_data   = d + p * dx;  // data offset
        float sum        = 0.0f;
        for(int i = 0; i < nx; ++i)
            sum += recon[i + pix_offset];
        float upd = (data[idx_data] - sum) / static_cast<float>(nx);
        for(int i = 0; i < nx; ++i)
            dst[i + pix_offset] += upd;
    }
}

//======================================================================================//

void
cuda_compute_projection(int dt, int dx, int nx, int ny, const float* theta, int s, int p,
                        int nthreads, gpu_data** _gpu_data)
{
    auto       thread_number = GetThisThreadID() % nthreads;
    gpu_data*& _cache        = _gpu_data[thread_number];

    cuda_set_device(_cache->device());
    //nppSetStream(0);

    // needed for recon to output at proper orientation
    float theta_p_rad = fmodf(theta[p] + halfpi, twopi);
    float theta_p_deg = theta_p_rad * degrees;
    int   block       = _cache->block();
    int   grid        = _cache->compute_grid(dx);

    const float* recon     = _cache->recon();
    const float* data      = _cache->data();
    float*       update    = _cache->update();
    float*       recon_rot = _cache->rot();
    float*       recon_tmp = _cache->tmp();

    // Rotate object
    hipMemset(recon_rot, 0, nx * ny * sizeof(float));
    cuda_rotate_ip(recon_rot, recon, -theta_p_rad, -theta_p_deg, nx, ny);
    hipMemcpy(recon_tmp, recon_rot, nx * ny * sizeof(float), hipMemcpyDeviceToDevice);

    NVTX_RANGE_PUSH(&nvtx_update);
    cuda_sirt_pixels_kernel<<<grid, block>>>(p, nx, dx, recon_rot, data, recon_tmp);
    NVTX_RANGE_POP(&nvtx_update);

    // Back-Rotate object
    hipMemcpy(recon_rot, recon_tmp, nx * ny * sizeof(float), hipMemcpyDeviceToDevice);
    hipMemset(recon_tmp, 0, nx * ny * sizeof(float));
    cuda_rotate_ip(recon_tmp, recon_rot, theta_p_rad, theta_p_deg, nx, ny);

    // update shared update array
    float factor = 1.0f / static_cast<float>(dx);
    cuda_sirt_atomic_sum_kernel<<<grid, block>>>(update, recon_tmp, nx * ny, factor);
}

//--------------------------------------------------------------------------------------//

void
sirt_cuda(const float* cpu_data, int dy, int dt, int dx, const float* center,
          const float* theta, float* cpu_recon, int ngridx, int ngridy, int num_iter)
{
    if(cuda_device_count() == 0)
        throw std::runtime_error("No CUDA device(s) available");

    cuda_device_query();

    printf("\n\t%s [nitr = %i, dy = %i, dt = %i, dx = %i, nx = %i, ny = %i]\n\n",
           __FUNCTION__, num_iter, dy, dt, dx, ngridx, ngridy);

    auto tid = GetThisThreadID();

    // assign the thread to a device
    set_this_thread_device();

    // get some properties
    int num_devices       = cuda_device_count();
    int nthreads          = GetEnv("TOMOPY_NUM_THREADS", HW_CONCURRENCY);
    nthreads              = std::max(nthreads, 1);
    hipStream_t* streams = create_streams(num_devices);

#if defined(TOMOPY_USE_PTL)
    TaskRunManager* run_man = cpu_run_manager();
    init_run_manager(run_man, nthreads);
    TaskManager* task_man = run_man->GetTaskManager();
    ThreadPool*  tp       = task_man->thread_pool();
#endif

    TIMEMORY_AUTO_TIMER("");

    // GPU allocated copies
    int master_device = 0;
    cuda_set_device(master_device);
    float* tmp_recon = gpu_malloc<float>(dy * ngridx * ngridy);
    float* recon = gpu_malloc<float>(dy * ngridx * ngridy);
    hipMemcpy(recon, cpu_recon, dy * ngridx * ngridy * sizeof(float),
               hipMemcpyHostToDevice);
    gpu_data** _gpu_data = new gpu_data*[nthreads];

    for(int ii = 0; ii < nthreads; ++ii)
        _gpu_data[ii] = new gpu_data(ii % num_devices, ii, dy, dt, dx, ngridx, ngridy, cpu_data);


    for(int i = 0; i < num_iter; i++)
    {
        START_TIMER(t_start);

        // set "update" to zero, copy in "recon"
        cuda_set_device(master_device);
        hipDeviceSynchronize();
        for(int ii = 0; ii < nthreads; ++ii)
        {
            _gpu_data[ii]->reset();
            _gpu_data[ii]->copy(recon);
        }

        // Loop over slices
        for(int s = 0; s < dy; ++s)
        {
#if defined(TOMOPY_USE_PTL)
            TaskGroup<void> tg;
            // For each projection angle
            for(int p = 0; p < dt; p++)
                task_man->exec(tg, cuda_compute_projection, dt, dx, ngridx, ngridy, theta,
                               s, p, nthreads, _gpu_data);
            tg.join();
#else
            // For each projection angle
            for(int p = 0; p < dt; p++)
                cuda_compute_projection(dt, dx, ngridx, ngridy, theta, s, p, nthreads,
                                        _gpu_data);
#endif
        }

        cuda_set_device(master_device);
        hipDeviceSynchronize();
        for(int ii = 0; ii < nthreads; ++ii)
        {
            int    dst_device = master_device;
            int    src_device = _gpu_data[ii]->m_device;
            int    block      = _gpu_data[ii]->block();
            int    grid       = _gpu_data[ii]->compute_grid(dy * ngridx * ngridy);
            float* src        = _gpu_data[ii]->update();
            float* dst        = src;
            if(src_device != dst_device)
            {
                cuda_set_device(src_device);
                hipDeviceSynchronize();
                cuda_set_device(master_device);
                dst = tmp_recon;
                hipMemcpyPeer(dst, dst_device, src, src_device, dy * ngridx * ngridy);
            }
            float factor = 1.0f;
            cuda_sirt_atomic_sum_kernel<<<grid, block>>>(recon, dst, dy * ngridx * ngridy,
                                                         factor);
        }
        hipDeviceSynchronize();
        REPORT_TIMER(t_start, "iteration", i, num_iter);
    }
    printf("\n");

    hipDeviceSynchronize();
    hipMemcpy(cpu_recon, recon, dy * ngridx * ngridy * sizeof(float),
               hipMemcpyDeviceToHost);
    hipFree(recon);
    hipFree(tmp_recon);

    for(int i = 0; i < nthreads; ++i)
        delete _gpu_data[i];
    delete[] _gpu_data;

    hipDeviceSynchronize();
    destroy_streams(streams, num_devices);
}

//======================================================================================//
