#include "hip/hip_runtime.h"
// Copyright (c) 2015, UChicago Argonne, LLC. All rights reserved.

// Copyright 2015. UChicago Argonne, LLC. This software was produced
// under U.S. Government contract DE-AC02-06CH11357 for Argonne National
// Laboratongridx (ANL), which is operated by UChicago Argonne, LLC for the
// U.S. Department of Energy. The U.S. Government has rights to use,
// reproduce, and distribute this software.  NEITHER THE GOVERNMENT NOR
// UChicago Argonne, LLC MAKES ANY WARRANTY, EXPRESS OR IMPLIED, OR
// ASSUMES ANY LIABILITY FOR THE USE OF THIS SOFTWARE.  If software is
// modified to produce derivative works, such modified software should
// be clearly marked, so as not to confuse it with the version available
// from ANL.

// Additionally, redistribution and use in source and binangridx forms, with
// or without modification, are permitted provided that the following
// conditions are met:

//     * Redistributions of source code must retain the above copyright
//       notice, this list of conditions and the following disclaimer.

//     * Redistributions in binangridx form must reproduce the above copyright
//       notice, this list of conditions and the following disclaimer in
//       the documentation and/or other materials provided with the
//       distribution.

//     * Neither the name of UChicago Argonne, LLC, Argonne National
//       Laboratongridx, ANL, the U.S. Government, nor the names of its
//       contributors may be used to endorse or promote products derived
//       from this software without specific prior written permission.

// THIS SOFTWARE IS PROVIDED BY UChicago Argonne, LLC AND CONTRIBUTORS
// "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
// LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
// FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL UChicago
// Argonne, LLC OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
// INCIDENTAL, SPECIAL, EXEMPLAngridx, OR CONSEQUENTIAL DAMAGES (INCLUDING,
// BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
// LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEOngridx OF LIABILITY, WHETHER IN CONTRACT, STRICT
// LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
// ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
// POSSIBILITY OF SUCH DAMAGE.

#include "common.hh"
#include "gpu.hh"
#include "utils.hh"
#include "utils_cuda.hh"

BEGIN_EXTERN_C
#include "mlem.h"
#include "utils.h"
#include "utils_openacc.h"
#include "utils_openmp.h"
END_EXTERN_C

#include <algorithm>
#include <chrono>
#include <cstdlib>
#include <memory>
#include <numeric>

#if defined(TOMOPY_USE_NVTX)
extern nvtxEventAttributes_t nvtx_total;
extern nvtxEventAttributes_t nvtx_iteration;
extern nvtxEventAttributes_t nvtx_slice;
extern nvtxEventAttributes_t nvtx_projection;
extern nvtxEventAttributes_t nvtx_update;
extern nvtxEventAttributes_t nvtx_rotate;
#endif

//======================================================================================//

__global__ void
cuda_mlem_pixels_kernel(int p, int nx, int dx, float* recon, const float* data,
                        const gpu_data::int_type* recon_use, float* sum_dist)
{
    int d0      = blockIdx.x * blockDim.x + threadIdx.x;
    int dstride = blockDim.x * gridDim.x;

    for(int d = d0; d < dx; d += dstride)
    {
        int fnx = 0;
        for(int i = 0; i < nx; ++i)
            sum_dist[d * nx + i] += recon[d * nx + i];
        for(int i = 0; i < nx; ++i)
            fnx += (recon_use[d * nx + i] != 0) ? 1 : 0;
        if(fnx != 0)
        {
            float sum = data[p * dx + d] / scast<float>(fnx);
            for(int i = 0; i < nx; ++i)
                recon[d * nx + i] += sum;
        }
    }
}

//======================================================================================//

__global__ void
cuda_mlem_update_kernel(float* recon, const float* update, const float* sum_dist,
                        int size)
{
    int i0      = blockIdx.x * blockDim.x + threadIdx.x;
    int istride = blockDim.x * gridDim.x;

    for(int i = i0; i < size; i += istride)
    {
        if(sum_dist[i] != 0.0f)
            recon[i] *= update[i] / sum_dist[i];
    }
}

//======================================================================================//

void
mlem_gpu_compute_projection(gpu_data::gpu_data_ptr_t _cache, int s, int p, int dy, int dt,
                            int dx, int nx, int ny, const float* theta)
{
    typedef gpu_data::int_type int_type;

#if defined(DEBUG)
    printf("[%lu] Running slice %i, projection %i on device %i...\n", GetThisThreadID(),
           s, p, _cache->device());
#endif

    // ensure running on proper device
    cuda_set_device(_cache->device());

    // calculate some values
    float        theta_p_rad = fmodf(theta[p] + halfpi, twopi);
    float        theta_p_deg = theta_p_rad * degrees;
    const float* recon       = _cache->recon() + s * nx * ny;
    const float* data        = _cache->data() + s * dt * dx;
    float*       update      = _cache->update() + s * nx * ny;
    float*       sum_dist    = _cache->sum_dist() + s * nx * ny;
    auto*        use_rot     = _cache->use_rot();
    auto*        use_tmp     = _cache->use_tmp();
    float*       rot         = _cache->rot();
    float*       tmp         = _cache->tmp();
    int          block       = _cache->block();
    int          grid        = _cache->compute_grid(nx);
    hipStream_t stream      = _cache->stream();

    gpu_memset<int_type>(use_rot, 0, nx * ny, stream);
    gpu_memset<float>(rot, 0, nx * ny, stream);
    gpu_memset<float>(tmp, 0, nx * ny, stream);

    // forward-rotate
    cuda_rotate_ip(use_rot, use_tmp, -theta_p_rad, -theta_p_deg, nx, ny, stream, GPU_NN);
    cuda_rotate_ip(rot, recon, -theta_p_rad, -theta_p_deg, nx, ny, stream);
    // compute simdata
    cuda_mlem_pixels_kernel<<<grid, block, 0, stream>>>(p, nx, dx, rot, data, use_rot,
                                                        sum_dist);
    // back-rotate
    cuda_rotate_ip(tmp, rot, theta_p_rad, theta_p_deg, nx, ny, stream);
    // update shared update array
    cuda_atomic_sum_kernel<<<grid, block, 0, stream>>>(update, tmp, nx * ny, 1.0f);
    // synchronize the stream (do this frequently to avoid backlog)
    stream_sync(stream);
}

//======================================================================================//

void
mlem_cuda(const float* cpu_data, int dy, int dt, int dx, const float* cpu_center,
          const float* theta, float* cpu_recon, int ngridx, int ngridy, int num_iter)
{
    typedef decltype(HW_CONCURRENCY) nthread_type;

    auto num_devices = cuda_device_count();
    if(num_devices == 0)
        throw std::runtime_error("No CUDA device(s) available");

    printf("[%lu]> %s : nitr = %i, dy = %i, dt = %i, dx = %i, nx = %i, ny = %i\n",
           GetThisThreadID(), __FUNCTION__, num_iter, dy, dt, dx, ngridx, ngridy);

    // initialize nvtx data
    init_nvtx();
    // print device info
    cuda_device_query();
    // thread counter for device assignment
    static std::atomic<int> ntid;

    // compute some properties (expected python threads, max threads, device assignment)
    auto min_threads   = nthread_type(1);
    auto pythreads     = GetEnv("TOMOPY_PYTHON_THREADS", HW_CONCURRENCY);
    auto max_threads   = HW_CONCURRENCY / std::max(pythreads, min_threads);
    auto nthreads      = std::max(GetEnv("TOMOPY_NUM_THREADS", max_threads), min_threads);
    int  thread_device = (ntid++) % num_devices;  // assign to device

#if defined(TOMOPY_USE_PTL)
    typedef TaskManager manager_t;
    TaskRunManager*     run_man = gpu_run_manager();
    init_run_manager(run_man, nthreads);
    TaskManager* task_man = run_man->GetTaskManager();
    ThreadPool*  tp       = task_man->thread_pool();
#else
    typedef void manager_t;
    void*        task_man = nullptr;
#endif

    TIMEMORY_AUTO_TIMER("");

    // GPU allocated copies
    cuda_set_device(thread_device);
    printf("[%lu] Running on device %i...\n", GetThisThreadID(), thread_device);

    typedef gpu_data::init_data_t      init_data_t;
    typedef gpu_data::gpu_data_array_t gpu_data_array_t;

    uintmax_t   recon_pixels = scast<uintmax_t>(dy * ngridx * ngridy);
    auto        block        = GetBlockSize();
    auto        grid         = ComputeGridSize(recon_pixels, block);
    auto        main_stream  = create_streams(1);
    float*      update    = gpu_malloc_and_memset<float>(recon_pixels, 0, *main_stream);
    float*      sum_dist  = gpu_malloc_and_memset<float>(recon_pixels, 0, *main_stream);
    init_data_t init_data = gpu_data::initialize(thread_device, nthreads, dy, dt, dx,
                                                 ngridx, ngridy, cpu_recon, cpu_data);
    gpu_data_array_t _gpu_data = std::get<0>(init_data);
    float*           recon     = std::get<1>(init_data);
    const float*     data      = std::get<2>(init_data);
    for(auto& itr : _gpu_data)
        itr->alloc_sum_dist();

    NVTX_RANGE_PUSH(&nvtx_total);

    for(int i = 0; i < num_iter; i++)
    {
        // timing and profiling
        TIMEMORY_AUTO_TIMER("");
        NVTX_RANGE_PUSH(&nvtx_iteration);
        START_TIMER(t_start);

        // sync and reset
        gpu_data::sync(_gpu_data);
        gpu_data::reset(_gpu_data);

        // execute
        execute<manager_t, gpu_data_array_t>(task_man, dy, dt, std::ref(_gpu_data),
                                             mlem_gpu_compute_projection, dy, dt, dx,
                                             ngridx, ngridy, theta);

        // sync the thread streams
        gpu_data::sync(_gpu_data);

        // sync the main stream
        stream_sync(*main_stream);

        // have threads add to global update and sum_dist
        for(auto& itr : _gpu_data)
        {
            auto nblock = itr->block();
            auto ngrid  = itr->compute_grid(recon_pixels);
            cuda_atomic_sum_kernel<<<ngrid, nblock, 0, itr->stream(0)>>>(update,
                                                                         itr->update(),
                                                                         recon_pixels,
                                                                         1.0f);
            cuda_atomic_sum_kernel<<<ngrid, nblock, 0, itr->stream(1)>>>(sum_dist,
                                                                         itr->sum_dist(),
                                                                         recon_pixels,
                                                                         1.0f);
        }

        // sync the thread streams
        gpu_data::sync(_gpu_data);

        // update the global recon with global update and sum_dist
        cuda_mlem_update_kernel<<<grid, block, 0, *main_stream>>>(recon, update, sum_dist,
                                                                  recon_pixels);

        // stop profile range and report timing
        NVTX_RANGE_POP(0);
        REPORT_TIMER(t_start, "iteration", i, num_iter);
    }

    // sync main stream
    stream_sync(*main_stream);

    // copy to cpu
    gpu2cpu_memcpy_and_free<float>(cpu_recon, recon, dy * ngridx * ngridy, 0);

    // ensure copy finished
    stream_sync(0);

    // destroy main stream
    destroy_streams(main_stream, 1);

    NVTX_RANGE_POP(0);
}

//======================================================================================//
